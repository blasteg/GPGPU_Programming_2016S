#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <vector>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void first_step(const char* text, int* count, int text_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx>=text_size)
		return;
	if (text[idx]>='a' && text[idx]<= 'z')
		count[idx]=1;
	else if (text[idx]>='A' && text[idx]<= 'Z')
		count[idx]=1;
	else
		count[idx]=0;
}

__global__ void count_up(int* input, int* output,int text_size,int IntendedValue,char* d_did_thing)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx>=text_size)
		return;
	if (idx<IntendedValue || input[idx]<IntendedValue)
		output[idx]=input[idx];
	else
	{
		output[idx]=input[idx]+input[idx-IntendedValue];
		*d_did_thing=(bool)1;
	}

}

__global__ void count_which_word(int* input,int* output,int text_size,int* head, int now_checking)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx>=text_size)
		return;
	if (idx<head[now_checking])
		output[idx]=input[idx];
	else
		output[idx]=input[idx]+1;
}

__global__ void Caesar_shift(char* text,int* pos,int* at_which_word,int text_size,int base_offset,int extra_offset_per_position,int extra_offset_per_word)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int letter,capital;
	if (idx>=text_size)
		return;
	if (text[idx]>='a' && text[idx]<= 'z')
	{
		letter=text[idx]-'a';
		capital=1;
	}
	else if (text[idx]>='A' && text[idx]<= 'Z')
	{
		letter=text[idx]-'A';
		capital=2;
	}
	else
		return;
	letter=letter+base_offset+pos[idx]*extra_offset_per_position;
	letter=letter%26;
	letter=letter++at_which_word[idx]*extra_offset_per_word;
	letter=letter%26;
	if (capital==1)
		text[idx]=letter+'a';
	if (capital==2)
		text[idx]=;etter+'A';
}

void CountPosition(const char *text, int *pos, int text_size)
{
	int gridSize=((text_size-1)/1024)+1;

	int i;
	int IntnededValue=1;
	first_step<<<gridSize, 1024>>>(text,pos,text_size);//first step: decide which are letter which are not
	char did_thing;
	char *d_did_thing;
	int *pos2;
	hipMalloc(&d_did_thing,sizeof(char));
	hipMalloc(&pos2,sizeof(int)*text_size);
	did_thing=1;
	i=0;
	while (did_thing)
	{
		i=i+1;
		did_thing=0;
		hipMemcpy(d_did_thing,&did_thing,sizeof(char),hipMemcpyHostToDevice);

		if (i%2)
			count_up<<<gridSize, 1024>>>(pos,pos2,text_size,IntnededValue,d_did_thing);
		else
			count_up<<<gridSize, 1024>>>(pos2,pos,text_size,IntnededValue,d_did_thing);
		IntnededValue=IntnededValue*2;
		
		hipMemcpy(&did_thing,d_did_thing,sizeof(char),hipMemcpyDeviceToHost);
		
	}
	if (i%2)
		hipMemcpy(pos,pos2,text_size*sizeof(int),hipMemcpyDeviceToDevice);
	hipFree(pos2);
	hipFree(d_did_thing);

	
}

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);
	std::vector<int> heads;

	// TODO
	thrust::device_ptr<const int> iter;
	int now=-1;
	while(now != (text_size-1))
	{
	iter= thrust::find ( pos_d+now+1,pos_d+text_size-1,1);
	now=iter-pos_d;
	if (*(pos_d+now) == 1)
	heads.push_back(now);
	//printf("%d/%d\n",now,text_size);
	}
	nhead=heads.size();
	//printf("%d\n",nhead);
	hipFree(buffer);
	hipMemcpy(head,heads.data(),nhead*sizeof(int),hipMemcpyHostToDevice);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
	int *at_which_word, *at_which_word_buffer,i;
	hipMalloc(&at_which_word, sizeof(int)*text_size);
	hipMalloc(&at_which_word_buffer, sizeof(int)*text_size);
	hipMemset (at_which_word, 0, sizeof(int)*text_size );
	hipMemset (at_which_word_buffer, 0, sizeof(int)*text_size );

	int gridSize=((text_size-1)/1024)+1;
	//counting the belonging of which word
	for (i=0;i<n_head;i++)
	{
		if (i%2==0)
			count_which_word<<<gridSize, 1024>>>(at_which_word,at_which_word_buffer,text_size,head,i);
		else
			count_which_word<<<gridSize, 1024>>>(at_which_word_buffer,at_which_word,text_size,head,i);
	}
	if(i%2)
		hipMemcpy(at_which_word,at_which_word_buffer,text_size*sizeof(int),hipMemcpyDeviceToDevice);
	hipFree(at_which_word_buffer);
	//My work : position-in-word andat-which-word dependent Caesar encoding.
	int base_offset=3;
	int extra_offset_per_position=1;
	int extra_offset_per_word=7;
	Caesar_shift<<<gridSize, 1024>>>(text,pos,at_which_word,text_size,base_offset,extra_offset_per_position,extra_offset_per_word);
}
