#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <vector>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void first_step(const char* text, int* count, int text_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx>=text_size)
		return;
	if (text[idx]>='a' && text[idx]<= 'z')
		count[idx]=1;
	else if (text[idx]>='A' && text[idx]<= 'Z')
		count[idx]=1;
	else
		count[idx]=0;
}

__global__ void count_up(int* input, int* output,int text_size,int IntendedValue)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx>=text_size)
		return;
	if (idx<IntendedValue || input[idx]<IntendedValue)
		output[idx]=input[idx];
	else
	{
		output[idx]=input[idx]+input[idx-IntendedValue];
	}

}

__global__ void count_which_word(int* output,int text_size,int* pos)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx>=text_size)
		return;
	if (pos[idx]==1)
		output[idx]=1;
	else
		output[idx]=0;
}

__global__ void Caesar_shift(char* text,int* pos,int* at_which_word,int text_size,int base_offset,int extra_offset_per_position,int extra_offset_per_word)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int letter,capital;
	if (idx>=text_size)
		return;
	if (text[idx]>='a' && text[idx]<= 'z')
	{
		letter=text[idx]-'a';
		capital=1;
	}
	else if (text[idx]>='A' && text[idx]<= 'Z')
	{
		letter=text[idx]-'A';
		capital=2;
	}
	else
		return;
	letter=letter+base_offset+pos[idx]*extra_offset_per_position;
	letter=letter%26;
	letter=letter+at_which_word[idx]*extra_offset_per_word;
	letter=letter%26;
	if (capital==1)
		text[idx]=letter+'a';
	if (capital==2)
		text[idx]=letter+'A';
}

void CountPosition(const char *text, int *pos, int text_size)
{
	int gridSize=((text_size-1)/1024)+1;

	int i;
	int IntnededValue=1;
	first_step<<<gridSize, 1024>>>(text,pos,text_size);//first step: decide which are letter which are not
	
	int *pos2;
	hipMalloc(&pos2,sizeof(int)*text_size);
	thrust::device_ptr<int> com_pos(pos), com_pos2(pos2);
	i=0;
	while (!(thrust::equal(thrust::device,com_pos,com_pos+text_size-1,com_pos2)))
	{
		i=i+1;
		
		if (i%2)
			count_up<<<gridSize, 1024>>>(pos,pos2,text_size,IntnededValue);
		else
			count_up<<<gridSize, 1024>>>(pos2,pos,text_size,IntnededValue);
		IntnededValue=IntnededValue*2;
		
	}
	if (i%2)
		hipMemcpy(pos,pos2,text_size*sizeof(int),hipMemcpyDeviceToDevice);
	hipFree(pos2);


	
}

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);

	// TODO
	hipMemcpy(buffer,pos,text_size*sizeof(int),hipMemcpyDeviceToDevice);
	thrust::device_vector<int> indices(text_size);
	thrust::sequence(indices.begin(), indices.end());
	nhead=thrust::count(pos_d,pos_d+text_size,1);
	int nzeros=thrust::count(pos_d,pos_d+text_size,0);
	thrust::sort_by_key(flag_d, flag_d+text_size, indices.begin());
	thrust::sort(indices.begin()+nzeros, indices.begin()+nzeros+nhead);
	thrust::copy(indices.begin()+nzeros, indices.begin()+nzeros+nhead, head_d);
	//printf("%d\n",nhead);
	hipFree(buffer);
	
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
	int *at_which_word;
	hipMalloc(&at_which_word, sizeof(int)*text_size);
	thrust::device_ptr<int> at_which_word_d(at_which_word);
	int gridSize=((text_size-1)/1024)+1;
	//counting the belonging of which word
	count_which_word<<<gridSize, 1024>>>(at_which_word,text_size,pos);
	//My work : position-in-word andat-which-word dependent Caesar encoding.
	thrust::inclusive_scan(at_which_word_d, at_which_word_d+text_size, at_which_word_d);
	int base_offset=3;
	int extra_offset_per_position=1;
	int extra_offset_per_word=7;
	Caesar_shift<<<gridSize, 1024>>>(text,pos,at_which_word,text_size,base_offset,extra_offset_per_position,extra_offset_per_word);
	hipFree(at_which_word);
}
