#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <vector>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void first_step(const char* text, int* count, int text_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx>=text_size)
		return;
	if (text[idx]>='a' && text[idx]<= 'z')
		count[idx]=1;
	else if (text[idx]>='A' && text[idx]<= 'Z')
		count[idx]=1;
	else
		count[idx]=0;
}

__global__ void count_up(int* input, int* output,int text_size,int IntendedValue,char* d_did_thing)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx>=text_size)
		return;
	if (idx<IntendedValue || input[idx]<IntendedValue)
		output[idx]=input[idx];
	else
	{
		output[idx]=input[idx]+input[idx-IntendedValue];
		*d_did_thing=(bool)1;
	}

}

void CountPosition(const char *text, int *pos, int text_size)
{
	int gridSize=((text_size-1)/32)+1;
	int i;
	int IntnededValue=1;
	first_step<<<gridSize, 32>>>(text,pos,text_size);//first step: decide which are letter which are not
	char did_thing;
	char *d_did_thing;
	int *pos2;
	hipMalloc(&d_did_thing,sizeof(char));
	hipMalloc(&pos2,sizeof(int)*text_size);
	did_thing=1;
	i=0;
	while (did_thing)
	{
		i=i+1;
		did_thing=0;
		hipMemcpy(d_did_thing,&did_thing,sizeof(char),hipMemcpyHostToDevice);
		if (i%2)
			count_up<<<gridSize, 32>>>(pos,pos2,text_size,IntnededValue,d_did_thing);
		else
			count_up<<<gridSize, 32>>>(pos2,pos,text_size,IntnededValue,d_did_thing);
		IntnededValue=IntnededValue*2;
		hipMemcpy(&did_thing,d_did_thing,sizeof(char),hipMemcpyDeviceToHost);
	}
	if (i%2)
		hipMemcpy(pos,pos2,sizeof(char),hipMemcpyDeviceToDevice);
	hipFree(pos2);
	hipFree(d_did_thing);

	
}

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);
	std::vector<int> heads;

	// TODO
	thrust::device_ptr<const int> iter;
	int now=0;
	while(now != text_size)
	{
	iter= thrust::find ( pos_d+now,pos_d+text_size,1);
	now=iter-pos_d;
	if (*(pos_d+now) == 1)
	heads.push_back(now);

	}
	nhead=heads.size();
	hipFree(buffer);
	hipMemcpy(head,heads.data(),nhead*sizeof(int),hipMemcpyHostToDevice);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
}
