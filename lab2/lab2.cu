#include "hip/hip_runtime.h"
#include "lab2.h"
#include "time.h"
#include <stdlib.h>

static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 10800;

int sign(int x) {
    return (x > 0) - (x < 0);
}
void init_board(int* sx, int* sy,int* x, int* y,int* lb, int* rb, int cei, int flr);
void show_boarder(uint8_t *yuv,int up, int down)
{
	hipMemset(yuv+W*up,255,W);
	hipMemset(yuv+W*down,255,W);
}
void show_ball(uint8_t *yuv,int x, int y)
{
	hipMemset(yuv+W*(y-1)+x-1,255,3);
	hipMemset(yuv+W*y+x-1,255,3);
	hipMemset(yuv+W*(y+1)+x-1,255,3);
}
void show_bouncer(uint8_t *yuv,int left_bar_xc,int left_bar_yc,int right_bar_xc,int right_bar_yc,int bar_length_perside,int bar_thickness)
{
	int i;
	for (i=0;i<H;i++)
	{
		if (i>=left_bar_yc-bar_length_perside && i<=left_bar_yc+bar_length_perside)
			hipMemset(yuv+W*i+left_bar_xc-bar_thickness,255,bar_thickness);
		if (i>=right_bar_yc-bar_length_perside && i<=right_bar_yc+bar_length_perside)
			hipMemset(yuv+W*i+right_bar_xc,255,bar_thickness);
	}
}
void ball_speed_up(int* vx,int*vy)
{
	int sgnx=sign(*vx);
	int sgny=sign(*vy);
	if (rand()%4>0)
		(*vx)=(*vx)+1*sgnx;
	if (sgny!=0)
	{
		if (rand()%4>2)
			(*vy)=(*vy)+1*sgny;
	}
	else
	{
		if (rand()%4>2)
			(*vy)=(*vy)+((rand()%2)*2-1);
	}

}
void print_number(uint8_t* yuv,int print_locationx,int print_locationy,int digit) /*20x30 digit*/
{
	int i;
	switch (digit)
	{
		case 0:
		for (i=0;i<30;i++)
		{
			if (i==0 || i==1 || i==28 || i==29)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else
			{
				hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,3);
				hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
			}
		}
		break;
		case 1:
		for (i=0;i<30;i++)
		{
				hipMemset(yuv+W*(print_locationy+i)+print_locationx+9,255,3);	
		}
		break;
		case 2:
		for (i=0;i<30;i++)
		{
			if (i==0 || i==1 || i==28 || i==29 || i==14 || i==15)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else if (i<14)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
			else
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,3);
		}
		break;
		case 3:
		for (i=0;i<30;i++)
		{
			if (i==0 || i==1 || i==28 || i==29 || i==14 || i==15)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else
			hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
		}
		break;
		case 4:
		for (i=0;i<30;i++)
		{
			if (i==14 || i==15)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else if (i<14)
			{
				hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,3);
				hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
			}
			else
				hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
		}
		break;
		case 5:
		for (i=0;i<30;i++)
		{
			if (i==0 || i==1 || i==28 || i==29 || i==14 || i==15)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else if (i<14)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,3);
			else
			hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
		}
		break;
		case 6:
		for (i=0;i<30;i++)
		{
			if (i==0 || i==1 || i==28 || i==29 || i==14 || i==15)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else if (i<14)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,3);
			else
			{
				hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,3);
				hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
			}
		}
		break;
		case 7:
		for (i=0;i<30;i++)
		{
			if (i==0 || i==1)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else
			hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
		}
		break;
		case 8:
		for (i=0;i<30;i++)
		{
			if (i==0 || i==1 || i==28 || i==29|| i==14 || i==15)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else
			{
				hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,3);
				hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
			}
		}
		break;
		case 9:
		for (i=0;i<30;i++)
		{
			if (i==0 || i==1 || i==28 || i==29|| i==14 || i==15)
			hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,20);
			else if (i<14)
			{
				hipMemset(yuv+W*(print_locationy+i)+print_locationx,255,3);
				hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
			}
			else
			hipMemset(yuv+W*(print_locationy+i)+print_locationx+17,255,3);
		}
		break;
		default:
		break;
	}
}

void show_score(uint8_t *yuv,int cei, int left_score, int right_score)
{
	int print_locationx=10;
	int print_locationy=cei+5;
	int temp=left_score;
	int digit_count=1;
	while(temp/10)
	{
		digit_count+=1;
		temp/=10;
	}
	temp=left_score;
	for (int i=digit_count;i>0;i--)
	{
		print_number(yuv,print_locationx,print_locationy,temp/(pow(10,i-1)));
		temp=temp%((int)(pow(10,i-1)));
		print_locationx+=22;
	}
	temp=right_score;
	print_locationx=W-10-22;
	digit_count=1;
	while(temp/10)
	{
		digit_count+=1;
		temp/=10;
	}
	temp=right_score;
	for (int i=digit_count;i>0;i--)
	{
		print_number(yuv,print_locationx,print_locationy,temp%10);
		temp=temp/10;
		print_locationx-=22;
	}
}



struct Lab2VideoGenerator::Impl {
	int left_bar_xc=10;
	int right_bar_xc=W-10;

	int bar_length_perside=15;
	int bar_thickness=2;
	int left_score=0;
	int right_score=0;
	int ball_x=W/2;
	
	int ball_xv=0;
	int ball_yv=0;
	int ceiling=10;
	int flor=H-40;
	int left_bar_yc=(flor+ceiling)/2;
	int right_bar_yc=(flor+ceiling)/2;
	int ball_y=(flor+ceiling)/2;
	int t=0;
	int bar_max_speed=3;
	char phase='i';
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 60;
	info.fps_d = 1;
};


void Lab2VideoGenerator::Generate(uint8_t *yuv) {
	hipMemset(yuv, 0, W*H);
	hipMemset(yuv+W*H, 128, W*H/2);
	if ((impl->phase)=='i')
	{
		init_board(&(impl->ball_xv),&(impl->ball_yv),&(impl->ball_x),&(impl->ball_y),&(impl->left_bar_yc),&(impl->right_bar_yc),(impl->ceiling),(impl->flor));
		(impl->phase)='r';
	}
	else if ((impl->phase)=='r')
	{
		if ((impl->ball_xv)>0)
		{
			int to_move=(impl->ball_y)-(impl->right_bar_yc);
			if (std::abs(to_move)>(impl->bar_max_speed))
				to_move=sign(to_move)*(impl->bar_max_speed);
			if (to_move>0 && ((impl->right_bar_yc)+(impl->bar_length_perside)+to_move)>(impl->flor))
				to_move=(impl->flor)-((impl->right_bar_yc)+(impl->bar_length_perside));
			if (to_move<0 && ((impl->right_bar_yc)-(impl->bar_length_perside)+to_move)<(impl->ceiling))
				to_move=(impl->ceiling)-((impl->right_bar_yc)-(impl->bar_length_perside));
			(impl->right_bar_yc)+=to_move;
		}
		else
		{
			int to_move=(impl->ball_y)-(impl->left_bar_yc);
			if (std::abs(to_move)>(impl->bar_max_speed))
				to_move=sign(to_move)*(impl->bar_max_speed);
			if (to_move>0 && ((impl->left_bar_yc)+(impl->bar_length_perside)+to_move)>(impl->flor))
				to_move=(impl->flor)-((impl->left_bar_yc)+(impl->bar_length_perside));
			if (to_move<0 && ((impl->left_bar_yc)-(impl->bar_length_perside)+to_move)<(impl->ceiling))
				to_move=(impl->ceiling)-((impl->left_bar_yc)-(impl->bar_length_perside));
			(impl->left_bar_yc)+=to_move;
		}
		(impl->ball_x)+=(impl->ball_xv);
		(impl->ball_y)+=(impl->ball_yv);
		if((impl->ball_y)<=(impl->ceiling))
		{
			(impl->ball_y)=(impl->ceiling)+((impl->ceiling)-(impl->ball_y))+1;
			(impl->ball_yv)=(-1)*(impl->ball_yv);
		}
		if((impl->ball_y)>=(impl->flor))
		{
			(impl->ball_y)=(impl->flor)-((impl->ball_y)-(impl->flor))-1;
			(impl->ball_yv)=(-1)*(impl->ball_yv);
		}
		if((impl->ball_x)<=(impl->left_bar_xc) && (impl->ball_y)>=(impl->left_bar_yc)-(impl->bar_length_perside) && (impl->ball_y)<=(impl->left_bar_yc)+(impl->bar_length_perside))
		{
			(impl->ball_x)=(impl->left_bar_xc)+((impl->left_bar_xc)-(impl->ball_x))+1;
			(impl->ball_xv)=(-1)*(impl->ball_xv);
			ball_speed_up(&(impl->ball_xv),&(impl->ball_yv));
		}
		if((impl->ball_x)>=(impl->right_bar_xc) && (impl->ball_y)>=(impl->right_bar_yc)-(impl->bar_length_perside) && (impl->ball_y)<=(impl->right_bar_yc)+(impl->bar_length_perside))
		{
			(impl->ball_x)=(impl->right_bar_xc)-((impl->ball_x)-(impl->right_bar_xc))-1;
			(impl->ball_xv)=(-1)*(impl->ball_xv);
			ball_speed_up(&(impl->ball_xv),&(impl->ball_yv));
		}
		if ((impl->ball_x)<=1)
		{
			(impl->ball_x)=1;
			(impl->phase)='g';
			(impl->t)=0;
		}
		if ((impl->ball_x)>=W-1)
		{
			(impl->ball_x)=W-1;
			(impl->phase)='g';
			(impl->t)=0;
		}
	}
	else if ((impl->phase)=='g')
	{
		if ((impl->t)==0)
		{
			if ((impl->ball_x)==1)
				(impl->right_score)+=1;
			if ((impl->ball_x)==(W-1))
				(impl->left_score)+=1;
		}
		if ((impl->t)%2)
		{
		hipMemset(yuv, 255, W*H);
		hipMemset(yuv+W*H, 128, W*H/2);
		}
		if((impl->t)==10)
			(impl->phase)='i';
		(impl->t)++;
	}
	
	show_boarder(yuv,(impl->ceiling),(impl->flor));
	show_score(yuv,(impl->flor),(impl->left_score),(impl->right_score));
	show_ball(yuv,(impl->ball_x),(impl->ball_y));
	show_bouncer(yuv,(impl->left_bar_xc),(impl->left_bar_yc),(impl->right_bar_xc),(impl->right_bar_yc),(impl->bar_length_perside),(impl->bar_thickness));
	
}




void init_board(int* sx, int* sy,int* x, int* y,int* lb, int* rb, int cei, int flr)
{
	srand(time(NULL));
	*sx=(rand()%2+1)*((rand()%2)*2-1);
	*sy=rand()%3-1;
	*x=W/2;
	*y=(flr+cei)/2;
	*lb=(flr+cei)/2;
	*rb=(flr+cei)/2;
}

