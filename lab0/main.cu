#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

__global__ void SomeTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int otheridx=fsize-1-idx;
	char buffer;
	if (idx>otheridx)
		return;
	if (input_gpu[idx] == '\n' || input_gpu[otheridx] != '\n')
		return;
	
	buffer=	input_gpu[idx];
	input_gpu[idx]=input_gpu[otheridx];
	input_gpu[otheridx]=buffer;
	
}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (not fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	int Nthreads=((fsize-1)/2)+1;
	int gridSize=((Nthreads-1)/32)+1;
	// My transform: flip the input text, except when one meets a changeline
	// Don't transform over the tail
	// And don't transform the line breaks
	SomeTransform<<<gridSize, 32>>>(input_gpu, fsize);

	puts(text_smem.get_cpu_ro());
	return 0;
}
